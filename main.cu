#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#define N 256

__global__ void fill_matrix(char *M) {
  int idx = threadIdx.x;
  M[idx] = 200;
}

int main() {
  char *M, *d_M;
  M = (char *)malloc(N * N * N * sizeof(char));
  hipMalloc((void **)&d_M, N * N * N * sizeof(char));
  hipMemcpy(d_M, M, N * N * N * sizeof(char), hipMemcpyHostToDevice);
  fill_matrix<<<1, N * N * N>>>(d_M);
  hipMemcpy(M, d_M, N * N * N * sizeof(char), hipMemcpyDeviceToHost);


  FILE *fp = fopen("animation.viz", "wb");

  return 0;
}